#include "hip/hip_runtime.h"
#include "Mandelbrot.h"

#include "MandelbrotMath.h"
#include <iostream>

#include "IndiceTools_GPU.h"



using gpu::IndiceTools;

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
extern __global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h, float t, DomaineMath domaineMath);
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

Mandelbrot::Mandelbrot(const Grid& grid, uint w, uint h, int N, uint n, const DomaineMath& domaineMath) :
	Animable_I<uchar4>(grid, w, h, "Mandelbrot_OMP_rgba_uchar4",domaineMath), variateurAnimation(Interval<int>(20, N), 1)
    {
    // Input
    this->n = n;
    this->N = N;

    // Tools
    this->t = 0;					// protected dans super classe Animable

    }

Mandelbrot::~Mandelbrot(void)
    {
    // rien
    }

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * Override
 */
void Mandelbrot::animationStep()
    {
    this->t = variateurAnimation.varierAndGet(); // in [0,2pi]
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/


/**
 * Override (code entrainement cuda)
 */
void Mandelbrot::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {

    Device::lastCudaError("rippling rgba uchar4 (before kernel)"); // facultatif, for debug only, remove for release

    mandelbrot<<<dg,db>>>(ptrDevPixels,w,h,t, domaineMath);

    Device::lastCudaError("rippling rgba uchar4 (after kernel)"); // facultatif, for debug only, remove for release

    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/



/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

