#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "IndiceTools_GPU.h"

#include "MandelbrotMath.h"
#include "DomaineMath_GPU.h"

using namespace gpu;

// Attention : 	Choix du nom est impotant!
//		VagueDevice.cu et non Vague.cu
// 		Dans ce dernier cas, probl�me de linkage, car le nom du .cu est le meme que le nom d'un .cpp (host)
//		On a donc ajouter Device (ou n'importequoi) pour que les noms soient diff�rents!

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h, float t, const DomaineMath& domaineMath);
__device__ void workPixel(uchar4* ptrColorIJ,int i, int j,const DomaineMath& domaineMath,MandelbrotMath* ptrMandelbrotMath, float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h, float t, const DomaineMath& domaineMath)
    {
    MandelbrotMath mandelbrotMath = MandelbrotMath(w, h);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;

    int s = TID; // Variable de travail
    int i, j;	// Les indices lignes et colonne

    while(s < WH)
	{
	    IndiceTools::toIJ(s, w, &i, &j); // s[0,W*H[ --> i[0,H[ j[0,W[
	    workPixel(&ptrDevPixels[s],i,j,domaineMath,&mandelbrotMath, t);
	    s += NB_THREAD;
	}

    }



/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/


    /**
     * i in [0,h[
     * j in [0,w[
     *
     * code commun a:
     * 	- entrelacementOMP
     * 	- forAutoOMP
     */
    __device__
    void workPixel(uchar4* ptrColorIJ,int i, int j,const DomaineMath& domaineMath,MandelbrotMath* ptrMandelbrotMath, float t)
        {
        // (i,j) domaine ecran dans N2
        // (x,y) domaine math dans R2

        double x;
        double y;
        domaineMath.toXY(i, j, &x, &y); // fill (x,y) from (i,j)

        //float t=variateurAnimation.get();

        ptrMandelbrotMath->colorXY(ptrColorIJ,x, y, t); // in [01]
        }


/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

