#include "MontecarloMultiGPU.h"

#include <iostream>

#include "Device.h"

// Besoin de la classe du précédent TP (monoGPU)
#include "Montecarlo.h"

#include "reductionADD.h" // Pour la réduction additive

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

#define PI 3.14159265358979323846264338327950288419716939937510

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

MontecarloMultiGPU::MontecarloMultiGPU(const Grid& grid, int nbFlechettes, float tolerance)
    {
    this->nbFlechettes = nbFlechettes;
    this->grid = grid;
    }

MontecarloMultiGPU::~MontecarloMultiGPU(void)
    {

    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

float MontecarloMultiGPU::run()
    {
    Device::lastCudaError("addVecteur (before)"); // temp debug

    //
    //working_kernel<<<dg,db, sizeOctet>>>(ptrDevGenerator, nbDart);


    int nbDevice=Device::getDeviceCount();
    int nbDartGPU = this->nbFlechettes / nbDevice;

    long sumTotal = 0;

    #pragma omp parallel for reduction(+:sumTotal)
    for(int idDevice=0; idDevice < nbDevice; idDevice++){

	hipSetDevice(idDevice); // Set du device

	Montecarlo montecarlo(this->grid, -1.00f, 1.00f, 1.00f, nbDartGPU); // MontecarloMultiGPU avec son nombre de fléchettes en fonction du nombre de devices (
	montecarlo.run();

	sumTotal += montecarlo.getCoutFlechette();

    }

    double piHat = sumTotal * 4 * 1.00f / nbFlechettes; // TODO sumTotal * height/nbFlechettes

    return piHat;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
